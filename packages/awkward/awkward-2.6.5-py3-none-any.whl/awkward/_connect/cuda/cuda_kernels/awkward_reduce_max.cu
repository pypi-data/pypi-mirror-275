#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

// BEGIN PYTHON
// def f(grid, block, args):
//     (toptr, fromptr, parents, lenparents, outlength, identity, invocation_index, err_code) = args
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_max_a", toptr.dtype, fromptr.dtype, parents.dtype]))(grid, block, (toptr, fromptr, parents, lenparents, outlength, identity, invocation_index, err_code))
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_max_b", toptr.dtype, fromptr.dtype, parents.dtype]))(grid, block, (toptr, fromptr, parents, lenparents, outlength, identity, invocation_index, err_code))
// out["awkward_reduce_max_a", {dtype_specializations}] = None
// out["awkward_reduce_max_b", {dtype_specializations}] = None
// END PYTHON

template <typename T, typename C, typename U>
__global__ void
awkward_reduce_max_a(
    T* toptr,
    const C* fromptr,
    const U* parents,
    int64_t lenparents,
    int64_t outlength,
    T identity,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < outlength) {
      toptr[thread_id] = identity;
    }
  }
}

template <typename T, typename C, typename U>
__global__ void
awkward_reduce_max_b(
    T* toptr,
    const C* fromptr,
    const U* parents,
    int64_t lenparents,
    int64_t outlength,
    T identity,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < lenparents) {
      C x = fromptr[thread_id];
      toptr[parents[thread_id]] =
          (x > toptr[parents[thread_id]] ? x : toptr[parents[thread_id]]);
    }
  }
}

#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

// BEGIN PYTHON
// def f(grid, block, args):
//     (toptr, fromptr, parents, lenparents, outlength, invocation_index, err_code) = args
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_argmin_a", toptr.dtype, fromptr.dtype, parents.dtype]))(grid, block, (toptr, fromptr, parents, lenparents, outlength, invocation_index, err_code))
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_argmin_b", toptr.dtype, fromptr.dtype, parents.dtype]))(grid, block, (toptr, fromptr, parents, lenparents, outlength, invocation_index, err_code))
// out["awkward_reduce_argmin_a", {dtype_specializations}] = None
// out["awkward_reduce_argmin_b", {dtype_specializations}] = None
// END PYTHON

template <typename T, typename C, typename U>
__global__ void
awkward_reduce_argmin_a(
    T* toptr,
    const C* fromptr,
    const U* parents,
    int64_t lenparents,
    int64_t outlength,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < outlength) {
      toptr[thread_id] = -1;
    }
  }
}

template <typename T, typename C, typename U>
__global__ void
awkward_reduce_argmin_b(
    T* toptr,
    const C* fromptr,
    const U* parents,
    int64_t lenparents,
    int64_t outlength,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < lenparents) {
      int64_t parent = parents[thread_id];
      if (toptr[parent] == -1 ||
          (fromptr[thread_id] < (fromptr[toptr[parent]]))) {
        toptr[parent] = thread_id;
      }
    }
  }
}

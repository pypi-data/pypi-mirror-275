#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

// BEGIN PYTHON
// def f(grid, block, args):
//     (toptr, fromptr, parents, lenparents, outlength, invocation_index, err_code) = args
//     atomicAdd_toptr = cupy.array(toptr, dtype=cupy.uint64)
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_sum_a", toptr.dtype, fromptr.dtype, parents.dtype]))(grid, block, (toptr, fromptr, parents, lenparents, outlength, atomicAdd_toptr, invocation_index, err_code))
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_sum_b", toptr.dtype, fromptr.dtype, parents.dtype]))(grid, block, (toptr, fromptr, parents, lenparents, outlength, atomicAdd_toptr, invocation_index, err_code))
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_sum_c", toptr.dtype, fromptr.dtype, parents.dtype]))(grid, block, (toptr, fromptr, parents, lenparents, outlength, atomicAdd_toptr, invocation_index, err_code))
// out["awkward_reduce_sum_a", {dtype_specializations}] = None
// out["awkward_reduce_sum_b", {dtype_specializations}] = None
// out["awkward_reduce_sum_c", {dtype_specializations}] = None
// END PYTHON

template <typename T, typename C, typename U>
__global__ void
awkward_reduce_sum_a(
    T* toptr,
    const C* fromptr,
    const U* parents,
    int64_t lenparents,
    int64_t outlength,
    uint64_t* atomicAdd_toptr,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < outlength) {
      atomicAdd_toptr[thread_id] = 0;
    }
  }
}

template <typename T, typename C, typename U>
__global__ void
awkward_reduce_sum_b(
    T* toptr,
    const C* fromptr,
    const U* parents,
    int64_t lenparents,
    int64_t outlength,
    uint64_t* atomicAdd_toptr,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < lenparents) {
      atomicAdd(atomicAdd_toptr + parents[thread_id],
                (uint64_t)fromptr[thread_id]);
    }
  }
}

template <typename T, typename C, typename U>
__global__ void
awkward_reduce_sum_c(
    T* toptr,
    const C* fromptr,
    const U* parents,
    int64_t lenparents,
    int64_t outlength,
    uint64_t* atomicAdd_toptr,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < outlength) {
      toptr[thread_id] = (T)atomicAdd_toptr[thread_id];
    }
  }
}

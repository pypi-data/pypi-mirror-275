#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

// BEGIN PYTHON
// def f(grid, block, args):
//     (toptr, fromptr, parents, lenparents, outlength, invocation_index, err_code) = args
//     atomicAdd_toptr = cupy.array(toptr, dtype=cupy.uint64)
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_count_64_a", toptr.dtype, parents.dtype]))(grid, block, (toptr, fromptr, parents, lenparents, outlength, atomicAdd_toptr, invocation_index, err_code))
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_count_64_b", toptr.dtype, parents.dtype]))(grid, block, (toptr, fromptr, parents, lenparents, outlength, atomicAdd_toptr, invocation_index, err_code))
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_reduce_count_64_c", toptr.dtype, parents.dtype]))(grid, block, (toptr, fromptr, parents, lenparents, outlength, atomicAdd_toptr, invocation_index, err_code))
// out["awkward_reduce_count_64_a", {dtype_specializations}] = None
// out["awkward_reduce_count_64_b", {dtype_specializations}] = None
// out["awkward_reduce_count_64_c", {dtype_specializations}] = None
// END PYTHON

template <typename T, typename U>
__global__ void
awkward_reduce_count_64_a(
    T* toptr,
    const bool* fromptr,
    const U* parents,
    int64_t lenparents,
    int64_t outlength,
    uint64_t* atomicAdd_toptr,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < outlength) {
      atomicAdd_toptr[thread_id] = 0;
    }
  }
}

template <typename T, typename U>
__global__ void
awkward_reduce_count_64_b(
    T* toptr,
    const bool* fromptr,
    const U* parents,
    int64_t lenparents,
    int64_t outlength,
    uint64_t* atomicAdd_toptr,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < lenparents) {
      atomicAdd(atomicAdd_toptr + parents[thread_id], (uint64_t)1);
    }
  }
}

template <typename T, typename U>
__global__ void
awkward_reduce_count_64_c(T* toptr,
                          const bool* fromptr,
                          const U* parents,
                          int64_t lenparents,
                          int64_t outlength,
                          uint64_t* atomicAdd_toptr,
                          uint64_t invocation_index,
                          uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < outlength) {
      toptr[thread_id] = (T)atomicAdd_toptr[thread_id];
    }
  }
}
